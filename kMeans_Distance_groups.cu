
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <bits/stdc++.h>
#include <stdlib.h>
#include <stdio.h>


#define HANDLE_CUDA(e) (HandleCudaError(e, __FILE__, __LINE__ ))

static int HandleCudaError(hipError_t e, const char *file, int line) {
    if (e != hipSuccess) {
        printf("CudaError: %s in %s at line %d\n", hipGetErrorString(e), file, line);
        return -1;
    } else {
    	return 0;
    }
}


using namespace std;


struct initialVariablesDoc
{
    int numDimsionsVector;
    int numDocuments;
    int numGroups;
};

initialVariablesDoc readDocumentForInitialVariables(int col, int row)
{

    int initialVariables[row];
    int i = 0;

    ifstream fin;
    int line;
    
    fin.open("documents/initialVariableDocuments.csv");
    fin>>line;
    
    while(!fin.eof()){
        initialVariables[i] = line;
        i += 1;

        fin>>line;
    }

    initialVariablesDoc result = {initialVariables[0],initialVariables[1],initialVariables[2]};

    return result;
}

int* readDocumentGroupGroundTruth(int col, int row)
{
    int* groupGroundTruth = new int[row];
    int i = 0;

    ifstream fin;
    int line;
    
    fin.open("documents/groupGroundTruth.csv");
    fin>>line;
    
    while(!fin.eof()){
        groupGroundTruth[i] = line;
        i += 1;

        fin>>line;
    }

    return groupGroundTruth;
}

float** readVectorDucment(int col, int row)
{
    float** vectorsDocuments = new float*[row];
    int i = 0;

    ifstream fin;
    string line;
    
    fin.open("documents/vectorsDocuments.csv");
    fin>>line;
    
    while(!fin.eof()){
        vectorsDocuments[i] = new float[col];

        // vectorsDocuments[i] = line;
        vectorsDocuments[i][0] = 5;
        vectorsDocuments[i][1] = 10;
        

        // cout<<"This is the line = "<<line<<endl;


        stringstream ss(line);
        int j=0;
        while (ss.good()) {
            string substr;
            getline(ss, substr, ',');
            // v.push_back(substr);
            vectorsDocuments[i][j] = stof(substr);
            // cout<<"hello = "<<substr<<endl;
            j++;
        }




        i += 1;

        fin>>line;
    }

    return vectorsDocuments;
}

void saveDocument(string file, int* groupGroundTruth,int numDocuments)
{
    std::ofstream myfile;
    myfile.open (file);
    

    for (int i = 0; i < numDocuments; ++i)
    {
        // cout<<groupGroundTruth[i]<<endl;
        myfile<<groupGroundTruth[i]<<endl;

    }
    myfile.close();
}

__global__ void averagePointGroups (int *groupGroundTruth, float *vectorsDocuments,float *sum, int *n,int numDocuments, int numDimsionsVector, int numGroups)
{
    int i; // This is for the Documents
    int j = threadIdx.x; // The Groups
    int k; // This is for the Vector


    for (k=0;k<numDimsionsVector;k++){
        sum[k+numDimsionsVector*j] = 0;
    }
    n[j] = 0;

    __syncthreads();


    for (i=0;i<numDocuments;i++)
    {
        if (groupGroundTruth[i] == j)
        {
            for (k=0;k<numDimsionsVector;k++){
                sum[k+numDimsionsVector*j] += vectorsDocuments[k+numDimsionsVector*i];
            }
            n[j]++;
        }
    }

    __syncthreads();


    for (k=0;k<numDimsionsVector;k++){
        sum[k+numDimsionsVector*j] = sum[k+numDimsionsVector*j]/n[j];
        // printf("xronia pola moro mou averagePoint = %d   %d  %d  %f  \n",k,j,n[j],sum[k+numDimsionsVector*j]);

    }




}


__global__ void calc_distance (float *vectorsDocuments,float *meanGroups,float *distance, int numDocuments, int numDimsionsVector, int numGroups)
{
    int i = blockIdx.x; // This is for the Documents
    int j = threadIdx.x; // The Groups
    int k; // This is for the Vector


    float total  = 0.0;
    float diff  = 0.0;
    for (k=0;k<numDimsionsVector;k++){
        diff = (vectorsDocuments[k+i*numDimsionsVector] - meanGroups[k+j*numDimsionsVector]);
        total += diff*diff;
    }
    distance[i+j*numDocuments]=total;

}


__global__ void findNewGroup (float *distance, int *groupGroundTruth, int numDocuments, int numDimsionsVector, int numGroups)
{
    int i= blockIdx.x; // This is for the Documents
    int j;  // The Groups
    // int k; // This is for the Vector


    float minDist;

    j = 0;
    minDist = distance[i+j*numDocuments];
    groupGroundTruth[i] = j;
    
    for (j=0;j<numGroups;j++)
    {
        // printf(" This is cool resutls = %d %d  %d  %d %f \n",i,j,i+j*numDocuments,numDocuments,distance[i+j*numDocuments]);
        if (minDist>distance[i+j*numDocuments])
        {   
            minDist = distance[i+j*numDocuments];
            groupGroundTruth[i] = j;
        } 
    }
    

}


int main(int argc, char** argv)
{

    // cout<<"This is the argv my friendo = "<<argv[1]<<endl;
    // ------------------------------ Read the Files ----------------------
    // initVar -> The initial Variables initVar.numDimsionsVector ...
    // groupGroundTruth -> The group Ground thruth in order to calcualte the average
    // vectorsDocuments-> The actual vercotrs of the documetns to calculate the average

    initialVariablesDoc initVar;
    initVar = readDocumentForInitialVariables(1,3);
    int* groupGroundTruth = readDocumentGroupGroundTruth(1,initVar.numDocuments);
    float** vectorsDocuments = readVectorDucment(initVar.numDimsionsVector,initVar.numDocuments);
    // ------------------------------ Read the Files ----------------------


    float *sum_d; // [vector,groups] -> summation of all the vector for Each Group 
    float sum_h[initVar.numGroups*initVar.numDimsionsVector];
    float sum_h_TEST[initVar.numGroups*initVar.numDimsionsVector];


    int *n_d; // [groups,1] -> number of documetns that we added in each group
    int n_h[initVar.numGroups];
    int n_h_TEST[initVar.numGroups];

    int *groupGroundTruth_d; 
    float *vectorsDocuments_d;


    float *distance_d; // [Documents,groups] -> summation of all the vector for Each Group 
    float distance_h[initVar.numDocuments*initVar.numGroups];



    // ---------------- Translate 2D to 1D ---------------- 
    float *vectorsDocuments1D = (float*)malloc(sizeof(float)*initVar.numDocuments*initVar.numDimsionsVector);
    int coun1DVec = 0;
    for (int i = 0; i < initVar.numDocuments; ++i)
    {
        for (int j = 0; j < initVar.numDimsionsVector; ++j)
        {
            vectorsDocuments1D[coun1DVec] = vectorsDocuments[i][j];
            coun1DVec++;
        }
    }

    // ---------------- Translate 2D to 1D ---------------- 

    // cout<<"This the number of Documents Each Group ----------- "<<endl;
    // for (int i = 0; i < 15; ++i)
    // {
    //     cout<<groupGroundTruth[i]<<endl;
    // }

    // ------------------ TEST the SUM --------------
    int i; // This is for the Documents
    int j; // The Groups
    int k; // This is for the Vector

    for (j=0;j<initVar.numGroups;j++)
    {
        for (k=0;k<initVar.numDimsionsVector;k++){
            sum_h_TEST[k+initVar.numDimsionsVector*j] = 0;
        }
        n_h_TEST[j] = 0;



        for (i=0;i<initVar.numDocuments;i++)
        {
            if (groupGroundTruth[i] == j)
            {
                for (k=0;k<initVar.numDimsionsVector;k++){
                    sum_h_TEST[k+initVar.numDimsionsVector*j] += vectorsDocuments1D[k+initVar.numDimsionsVector*i];
                }
                n_h_TEST[j]++;
            }
        }


        for (k=0;k<initVar.numDimsionsVector;k++){
            sum_h_TEST[k+initVar.numDimsionsVector*j] = sum_h_TEST[k+initVar.numDimsionsVector*j]/n_h_TEST[j];

        }

    }
    cout<<" The test of the HOST for the sum -----------"<<endl;
    for (int i = 0; i < initVar.numGroups; ++i)
    {
        for (int j = 0; j < initVar.numDimsionsVector; ++j)
        {
            cout<<sum_h_TEST[j+i*initVar.numDimsionsVector]<<" ";
        }
        cout<<endl;
    }
    // ---------------------------------------------------


    // ---------------- cudaMalloc -------------
    HANDLE_CUDA(hipMalloc(( void **) &sum_d,initVar.numGroups*initVar.numDimsionsVector*sizeof(float)));
    HANDLE_CUDA(hipMalloc(( void **) &sum_h,initVar.numGroups*initVar.numDimsionsVector*sizeof(float)));

    HANDLE_CUDA(hipMalloc(( void **) &n_d,initVar.numGroups*sizeof(int)));
    HANDLE_CUDA(hipMalloc(( void **) &n_h,initVar.numGroups*sizeof(int)));

    HANDLE_CUDA(hipMalloc(( void **) &distance_d,initVar.numDocuments*initVar.numGroups*sizeof(float)));
    HANDLE_CUDA(hipMalloc(( void **) &distance_h,initVar.numDocuments*initVar.numGroups*sizeof(float)));
    

    HANDLE_CUDA(hipMalloc(( void **) &groupGroundTruth_d,initVar.numDocuments*sizeof(int)));
    HANDLE_CUDA(hipMalloc(( void **) &vectorsDocuments_d,initVar.numDocuments*initVar.numDimsionsVector*sizeof(float)));
    // ---------------- cudaMalloc -------------

    // ---------------- cudaMemcpy  - HostToDevice -------------
    HANDLE_CUDA(hipMemcpy(groupGroundTruth_d,groupGroundTruth,initVar.numDocuments*sizeof(int),hipMemcpyHostToDevice));
    HANDLE_CUDA(hipMemcpy(vectorsDocuments_d,vectorsDocuments1D,initVar.numDocuments*initVar.numDimsionsVector*sizeof(float),hipMemcpyHostToDevice));

    HANDLE_CUDA(hipMemcpy(sum_d,sum_h,initVar.numGroups*initVar.numDimsionsVector*sizeof(float),hipMemcpyHostToDevice));

    HANDLE_CUDA(hipMemcpy(n_d,n_h,initVar.numGroups*sizeof(int),hipMemcpyHostToDevice));

    HANDLE_CUDA(hipMemcpy(distance_d,distance_h,initVar.numDocuments*initVar.numGroups*sizeof(float),hipMemcpyHostToDevice));

    // ---------------- cudaMemcpy  - HostToDevice  -------------


    // ---------------- kMeans Algorithm on CUDA code -------------
    int iter;
    long arg = strtol(argv[1], NULL, 10);
    for (iter = 0; iter<int(arg);iter++)
    {
        averagePointGroups<<<1,initVar.numGroups>>>(groupGroundTruth_d,vectorsDocuments_d,sum_d,n_d,initVar.numDocuments,initVar.numDimsionsVector,initVar.numGroups);

        calc_distance<<<initVar.numDocuments,initVar.numGroups>>>(vectorsDocuments_d,sum_d,distance_d,initVar.numDocuments,initVar.numDimsionsVector,initVar.numGroups);

        findNewGroup<<<initVar.numDocuments,1>>>(distance_d,groupGroundTruth_d,initVar.numDocuments,initVar.numDimsionsVector,initVar.numGroups);

    }
    // ---------------- kMeans Algorithm on CUDA code -------------



    // ---------------- cudaMemcpy  - DeviceToHost  -------------
    HANDLE_CUDA(hipMemcpy(sum_h,sum_d,initVar.numGroups*initVar.numDimsionsVector*sizeof(float),hipMemcpyDeviceToHost));
    HANDLE_CUDA(hipMemcpy(n_h,n_d,initVar.numGroups*sizeof(int),hipMemcpyDeviceToHost));

    HANDLE_CUDA(hipMemcpy(distance_h,distance_d,initVar.numDocuments*initVar.numGroups*sizeof(float),hipMemcpyDeviceToHost));

    HANDLE_CUDA(hipMemcpy(groupGroundTruth,groupGroundTruth_d,initVar.numDocuments*sizeof(float),hipMemcpyDeviceToHost));
    // ---------------- cudaMemcpy  - DeviceToHost  -------------

    cout.setf(ios::fixed,ios::floatfield);
    cout.precision(3);



    cout<<"This the Average of Each Group -----------"<<endl;
    for (int i = 0; i < initVar.numGroups; ++i)
    {
        for (int j = 0; j < initVar.numDimsionsVector; ++j)
        {
            cout<<sum_h[j+i*initVar.numDimsionsVector]<<" ";
        }
        cout<<endl;
    }

    


    cout<<"This the number of Documents Each Group ----------- "<<endl;
    // for (int i = 0; i < 4; ++i)
    // {
    //     cout<<groupGroundTruth[i]<<endl;
    // }

    saveDocument("documents/groupGroundTruth2.csv",groupGroundTruth,initVar.numDocuments);

    // cout<<"Group of Every Document ----------- "<<endl;
    // for (int i = 0; i < initVar.numGroups; ++i)
    // {
       
    //     cout<<n_h[i]<<endl;

    // }

    // cout.setf(ios::fixed,ios::floatfield);
    // cout.precision(1);

    cout<<"This the Average of Each Group ----------- "<<endl;
    // for (int i = 0; i < 2; ++i)
    for (int i = 0; i < initVar.numGroups; ++i)
    {
        for (int j = 0; j < initVar.numDocuments; ++j)
        // for (int j = 0; j < 2; ++j)
        {
            cout<<distance_h[j+i*initVar.numDocuments]<<" ";
        }
        cout<<endl;
    }


    // ------------ Free Memory -----------
    // free(sum_h);
    // free(n_h);
    // free(distance_h);
    // free(groupGroundTruth);


    // HANDLE_CUDA(cudaFree(sum_d));
    // HANDLE_CUDA(cudaFree(n_d));
    // HANDLE_CUDA(cudaFree(distance_d));
    // HANDLE_CUDA(cudaFree(groupGroundTruth_d));
    // HANDLE_CUDA(cudaFree(vectorsDocuments_d));
    // HANDLE_CUDA(cudaFree(sum_d));

    // ------------ Free Memory -----------




    return 0;
}